
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <cstddef>
#include <iomanip>
#include <iostream>

constexpr size_t N = 8192;
auto constexpr BLK_N = 32;

static inline bool feq(float a, float b) { return abs(a - b) <= 1e-5f; }
// use macro for both host and device
#define at(arr, i, j) ((arr)[(i)*N + (j)])
#define L(a, b, n) ((a) * (n) + (b))

// Kernel function to add the elements of two arrays
__global__ void matmul(float *C, float *A, float *B) {
  __shared__ float sC[BLK_N][BLK_N];
  __shared__ float sA[BLK_N][BLK_N];
  __shared__ float sB[BLK_N][BLK_N];

  // all the indices are named as ijk, where:
  //    i: means who use, t for thread, b for block
  //    j: means related to which matrix, m for all matrix, b for block matrix
  //    k: means related to which axis, x for x axis, y for y axis, c for count

  auto const ty = threadIdx.y;
  auto const tx = threadIdx.x;
  auto const by = blockIdx.y;
  auto const bx = blockIdx.x;

  sC[ty][tx] = 0.0f;

  for (auto bk = 0; bk < N / BLK_N; bk++) {
    sA[ty][tx] = at(A, by * BLK_N + ty, bk * BLK_N + tx);
    sB[ty][tx] = at(B, bk * BLK_N + ty, bx * BLK_N + tx);

    __syncthreads();

    for (auto tk = 0; tk < BLK_N; tk++) {
      sC[ty][tx] += sA[ty][tk] * sB[tk][tx];
    }

    __syncthreads();
  }

  at(C, by * BLK_N + ty, bx * BLK_N + tx) = sC[ty][tx];
}

// host copies of A, B, C
float hA[N * N], hB[N * N], hC[N * N];

auto test() {
  // init two matrixes
  for (auto y = 0; y < N; y++) {
    for (auto x = 0; x < N; x++) {
      at(hA, y, x) = y == x ? 1.0f : 0.0f;
    }
  }

  for (auto y = 0; y < N; y++) {
    for (auto x = 0; x < N; x++) {
      at(hB, y, x) = y == x ? 1.0f : 0.0f;
    }
  }

  // device copies of A, B, C
  float *A, *B, *C;

  // Allocate Unified Memory – accessible from CPU or GPU
  auto constexpr MAT_SIZE = N * N * sizeof(float);
  hipMallocManaged(&A, MAT_SIZE);
  hipMallocManaged(&B, MAT_SIZE);
  hipMallocManaged(&C, MAT_SIZE);

  hipMemcpy(A, hA, MAT_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(B, hB, MAT_SIZE, hipMemcpyHostToDevice);
  hipMemset(C, 1, MAT_SIZE);

  dim3 constexpr blockDim{BLK_N, BLK_N, 1};
  dim3 constexpr gridDim{N / BLK_N, N / BLK_N, 1};

  auto const matmul_begin = std::chrono::steady_clock::now();

  matmul<<<gridDim, blockDim>>>(C, A, B);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  auto const matmul_end = std::chrono::steady_clock::now();

  hipMemcpy(hC, C, MAT_SIZE, hipMemcpyDeviceToHost);

  // Check for errors
  for (auto i = 0; i < N; i++) {
    for (auto j = 0; j < N; j++) {
      auto const v = at(C, i, j);
      if ((i == j && !feq(v, 1.0f)) || (i != j && !feq(v, 0.0f))) {
        std::cout << "Error at (" << i << ", " << j << "): " << v << std::endl;
        std::exit(EXIT_FAILURE);
      }
    }
  }

  // Free memory
  hipFree(A);
  hipFree(B);
  hipFree(C);

  auto const matmul_time =
      std::chrono::duration_cast<std::chrono::milliseconds>(matmul_end -
                                                            matmul_begin)
          .count();

  return matmul_time;
}

int main(void) {
  auto constexpr WARMUP_N = 5;
  auto constexpr TEST_N = 30;

  for (auto i = 0; i < WARMUP_N; i++) {
    test();
  }

  auto matmul_time = 0.0;
  for (auto i = 0; i < TEST_N; i++) {
    matmul_time += test();
  }
  matmul_time /= TEST_N;

  // each cell needs N fma, and there are N * N cells
  auto const matmul_tflops = 2.0 * N * N * N / matmul_time / 1e9;
  auto constexpr theory_max_tflops = 14.7456;
  auto constexpr cublas_tflops = 8.228;

  std::cout << std::setprecision(3) << std::fixed;
  std::cout << "matmul time: " << matmul_time << " ms" << std::endl;
  std::cout << "Throughput: " << matmul_tflops << " TFLOPS \n"
            << "    (" << matmul_tflops / theory_max_tflops * 100 << "% Max)\n"
            << "    (" << matmul_tflops / cublas_tflops * 100 << "% cuBLAS)\n"
            << std::endl;

  return 0;
}

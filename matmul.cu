
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <cstddef>
#include <hipblas.h>
#include <iomanip>
#include <iostream>
#include <random>

// just a constant
auto constexpr THREAD_PRE_WARP = 32;

// how many elements (float)
auto constexpr N = 8192;
auto constexpr M = 8192;
auto constexpr K = 8192;
// a N*K @ K*M matrix-matrix multiplication

//=================================================//
// how many elements in a thread-tile
auto constexpr TN = 2;
auto constexpr TM = 4;

// how to place thread-tiles in a warp-tile
auto constexpr TN_CNT = 8;
auto constexpr TM_CNT = 4;

// how to place warp-tiles in a block-tile
auto constexpr WN_CNT = 2;
auto constexpr WM_CNT = 2;

// how many elements should a block-tile have
auto constexpr BN = 64;
auto constexpr BM = 64;
auto constexpr BK = 32;
//=================================================//

// warp-tile size
auto constexpr WN = TN * TN_CNT;
auto constexpr WM = TM * TM_CNT;
static_assert(BN % WN == 0 && BK % WN == 0, "Unaligned WN");
static_assert(BM % WM == 0 && BK % WM == 0, "Unaligned WM");

struct DebugInfo {
  int no;
  float left, right;
  int by, bx, bk;
  int y, x, k;
  dim3 threadIdx;
  dim3 blockIdx;
};

std::ostream &operator<<(std::ostream &os, const dim3 &d) {
  os << "(" << d.x << ", " << d.y << ", " << d.z << ")";
  return os;
}

void print_debug_info(const DebugInfo &info) {
  std::cout << "no: " << info.no << std::endl;
  std::cout << "left: " << info.left << std::endl;
  std::cout << "right: " << info.right << std::endl;
  std::cout << "by: " << info.by << std::endl;
  std::cout << "bx: " << info.bx << std::endl;
  std::cout << "bk: " << info.bk << std::endl;
  std::cout << "y: " << info.y << std::endl;
  std::cout << "x: " << info.x << std::endl;
  std::cout << "k: " << info.k << std::endl;
  std::cout << "threadIdx: " << info.threadIdx << std::endl;
  std::cout << "blockIdx: " << info.blockIdx << std::endl;
}

__global__ void matmul(float *C, float *A, float *B, DebugInfo *debug);
void run(float *dC, float *dA, float *dB) {
  DebugInfo *deviceDebug;
  hipMalloc(&deviceDebug, sizeof(*deviceDebug));
  hipMemset(deviceDebug, 0, sizeof(*deviceDebug));

  auto constexpr THREAD_PRE_BLK = THREAD_PRE_WARP * WN_CNT * WM_CNT;
  auto constexpr GN = N / BN, GM = M / BM;

  dim3 constexpr blockDim{THREAD_PRE_BLK, 1, 1};
  dim3 constexpr gridDim{GN, GM, 1};
  matmul<<<gridDim, blockDim>>>(dC, dA, dB, deviceDebug);

  DebugInfo hostDebug;
  hipMemcpy(&hostDebug, deviceDebug, sizeof(*deviceDebug),
             hipMemcpyDeviceToHost);
  hipFree(deviceDebug);

  if (hostDebug.no != 0) {
    print_debug_info(hostDebug);
    std::exit(EXIT_FAILURE);
  }
}

// Kernel function to add the elements of two arrays
__global__ void matmul(float *C, float *A, float *B, DebugInfo *debug) {
  debug->no = -1; // if -1, the function return unexpectedly
  debug->threadIdx = threadIdx;
  debug->blockIdx = blockIdx;

  // each block has some smem cache
  __shared__ float sAr[BK][BN];
  __shared__ float sB[BK][BM];

  // each thread has some register cache
  float pA[TN], pB[TM];
  float pC[TN][TM];

  for (auto bk_idx = 0; bk_idx < K / BK; bk_idx++) {
    // implicitly, we have two by/bx for loops here
    auto const by_idx = blockIdx.y;
    auto const bx_idx = blockIdx.x;

    auto const by = by_idx * BN;
    auto const bx = bx_idx * BM;
    auto const bk = bk_idx * BK;

    // implicitly, we have two wy/wx for loops here
    auto const warp_id = threadIdx.x / THREAD_PRE_WARP;
    auto const wy_idx = warp_id / WM_CNT;
    auto const wx_idx = warp_id % WM_CNT;

    // implicitly, we have two ty/tx for loops here
    auto const thread_id = threadIdx.x % THREAD_PRE_WARP;
    auto const ty_idx = thread_id / TM_CNT;
    auto const tx_idx = thread_id % TM_CNT;

    auto const ty = ty_idx * TN;
    auto const tx = tx_idx * TM;

    { // load A to sA
      auto const ITER_Y = BN / (WN_CNT * WN), ITER_X = BK / (WM_CNT * WM);
      for (auto iter_y = 0; iter_y < ITER_Y; iter_y++) {
        for (auto iter_x = 0; iter_x < ITER_X; iter_x++) {
          auto const wy = (iter_y * WN_CNT + wy_idx) * WN;
          auto const wx = (iter_x * WM_CNT + wx_idx) * WM;

          for (auto ey = 0; ey < TN; ey++) {
            for (auto ex = 0; ex < TM; ex++) {
              auto const sy = wy + ty + ey;
              auto const sx = wx + tx + ex;

              sAr[sx][sy] = A[(by + sy) * K + (bk + sx)];
            }
          }
        }
      }
    }

    { // load B to sB
      auto const ITER_Y = BK / (WN_CNT * WN), ITER_X = BM / (WM_CNT * WM);
      for (auto iter_y = 0; iter_y < ITER_Y; iter_y++) {
        for (auto iter_x = 0; iter_x < ITER_X; iter_x++) {
          auto const wy = (iter_y * WN_CNT + wy_idx) * WN;
          auto const wx = (iter_x * WM_CNT + wx_idx) * WM;

          for (auto ey = 0; ey < TN; ey++) {
            for (auto ex = 0; ex < TM; ex++) {
              auto const sy = wy + ty + ey;
              auto const sx = wx + tx + ex;

              sB[sy][sx] = B[(bk + sy) * M + (bx + sx)];
            }
          }
        }
      }
    }

    __syncthreads();

    // check whether sA & sB is loaded correctly
    if constexpr (false) {
      for (auto y = 0; y < BN; y++) {
        for (auto k = 0; k < BK; k++) {
          if (sAr[k][y] != A[(by + y) * K + (bk + k)]) {
            debug->no = 1;
            debug->left = sAr[k][y];
            debug->right = A[(by + y) * K + (bk + k)];
            debug->by = by;
            debug->bx = bx;
            debug->bk = bk;
            debug->y = y;
            debug->k = k;
            return;
          }
        }
      }

      for (auto k = 0; k < BK; k++) {
        for (auto x = 0; x < BM; x++) {
          if (sB[k][x] != B[(bk + k) * M + (bx + x)]) {
            debug->no = 2;
            return;
          }
        }
      }
    }

    { // compute C
      auto const ITER_Y = BN / (WN_CNT * WN), ITER_X = BM / (WM_CNT * WM);
      for (auto iter_y = 0; iter_y < ITER_Y; iter_y++) {
        for (auto iter_x = 0; iter_x < ITER_X; iter_x++) {
          auto const wy = (iter_y * WN_CNT + wy_idx) * WN;
          auto const wx = (iter_x * WM_CNT + wx_idx) * WM;

          // clear pC
          for (auto ey = 0; ey < TN; ey++) {
            for (auto ex = 0; ex < TM; ex++) {
              pC[ey][ex] = 0.0f;
            }
          }

          for (auto k = 0; k < BK; k++) {
            // load pA
            for (auto ey = 0; ey < TN; ey++) {
              auto const sy = wy + ty + ey;
              pA[ey] = sAr[k][sy];
            }
            // load pB
            for (auto ex = 0; ex < TM; ex++) {
              auto const sx = wx + tx + ex;
              pB[ex] = sB[k][sx];
            }
            // compute
            for (auto ey = 0; ey < TN; ey++) {
              for (auto ex = 0; ex < TM; ex++) {
                pC[ey][ex] += pA[ey] * pB[ex];
              }
            }
          }

          // store pC
          for (auto ey = 0; ey < TN; ey++) {
            for (auto ex = 0; ex < TM; ex++) {
              auto const sy = wy + ty + ey;
              auto const sx = wx + tx + ex;
              C[(by + sy) * M + (bx + sx)] += pC[ey][ex];
            }
          }
        }
      }
    }

    __syncthreads();
  }

  debug->no = 0;
  return;
}

static inline bool feq(float a, float b) { return abs(a - b) <= 1e-2f; }

// host copies of A, B, C
float hA[N * K], hB[K * M], hC[N * M], std_hC[N * M];
float *dA, *dB, *dC;

static inline float randf() {
  static std::random_device rd{};
  static std::mt19937 e{rd()};
  static std::uniform_real_distribution<float> d{-1.0f, 1.0f};
  return d(e);
}

void mat_random(float *M, size_t n, size_t m) {
  for (auto y = 0; y < n; y++) {
    for (auto x = 0; x < m; x++) {
      M[y * m + x] = randf();
    }
  }
}

bool mat_eq(float *A, float *B, size_t n, size_t m) {
  for (auto y = 0; y < n; y++) {
    for (auto x = 0; x < m; x++) {
      if (!feq(A[y * m + x], B[y * m + x])) {
        std::cout << "A[" << y << "][" << x << "] = " << A[y * m + x]
                  << " != B[" << y << "][" << x << "] = " << B[y * m + x]
                  << std::endl;
        return false;
      }
    }
  }
  return true;
}

void calc_std() {
  float *dA, *dB, *dC;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float alpha = 1.0f;
  float beta = 0.0f;

  hipMalloc(&dA, sizeof(hA));
  hipMemcpy(dA, hA, sizeof(hA), hipMemcpyHostToDevice);

  hipMalloc(&dB, sizeof(hB));
  hipMemcpy(dB, hB, sizeof(hB), hipMemcpyHostToDevice);

  hipMalloc(&dC, sizeof(hC));
  hipMemset(dC, 0, sizeof(hC));
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dB, N, dA, K,
              &beta, dC, N);
  hipMemcpy(std_hC, dC, sizeof(hC), hipMemcpyDeviceToHost);

  hipblasDestroy(handle);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
}

void init() {
  mat_random(hA, N, K);
  mat_random(hB, K, M);
  calc_std();
}

auto test() {
  // device copies of A, B, C
  float *A, *B, *C;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMalloc(&A, sizeof(hA));
  hipMalloc(&B, sizeof(hB));
  hipMalloc(&C, sizeof(hC));

  hipMemcpy(A, hA, sizeof(hA), hipMemcpyHostToDevice);
  hipMemcpy(B, hB, sizeof(hB), hipMemcpyHostToDevice);
  hipMemset(C, 0, sizeof(hC));

  auto const matmul_begin = std::chrono::steady_clock::now();
  run(C, A, B);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  auto const matmul_end = std::chrono::steady_clock::now();

  hipMemcpy(hC, C, sizeof(hC), hipMemcpyDeviceToHost);

  // Check for errors
  if (!mat_eq(hC, std_hC, N, M)) {
    std::cout << "Error!" << std::endl;
    std::exit(EXIT_FAILURE);
  }

  // Free memory
  hipFree(A);
  hipFree(B);
  hipFree(C);

  using std::chrono::duration_cast;
  using std::chrono::milliseconds;
  return duration_cast<milliseconds>(matmul_end - matmul_begin).count();
}

int main(void) {
  init();
  std::cout << "Finish init" << std::endl;

#ifndef PROFILE
  auto constexpr WARMUP_N = 5;
  auto constexpr TEST_N = 30;

  for (auto i = 0; i < WARMUP_N; i++) {
    test();
  }

  auto matmul_time = 0.0;
  for (auto i = 0; i < TEST_N; i++) {
    matmul_time += test();
  }
  matmul_time /= TEST_N;
#else
  auto const matmul_time = test();
#endif

  // each cell needs N fma, and there are N * N cells
  auto const matmul_tflops = 2.0 * N * M * K / matmul_time / 1e9;
  auto constexpr theory_max_tflops = 14.7456;
  auto constexpr cublas_tflops = 8.228;

  std::cout << std::setprecision(3) << std::fixed;
  std::cout << "matmul time: " << matmul_time << " ms" << std::endl;
  std::cout << "Throughput: " << matmul_tflops << " TFLOPS \n"
            << "    (" << matmul_tflops / theory_max_tflops * 100 << "% Max)\n"
            << "    (" << matmul_tflops / cublas_tflops * 100 << "% cuBLAS)\n"
            << std::endl;

  return 0;
}

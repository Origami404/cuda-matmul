
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <cstddef>
#include <iomanip>
#include <iostream>

constexpr size_t N = 8192;
// how many elements a thread handles (THD_N * THD_N)
auto constexpr THD_N = 8;
// how many threads in a block
auto constexpr BLK_N = 8;

static inline bool feq(float a, float b) { return abs(a - b) <= 1e-5f; }
// use macro for both host and device
#define at(arr, i, j) ((arr)[(i)*N + (j)])

// Kernel function to add the elements of two arrays
__global__ void matmul(float *C, float *A, float *B) {
  auto const tx_beg = (blockIdx.x * BLK_N + threadIdx.x) * THD_N;
  auto const ty_beg = (blockIdx.y * BLK_N + threadIdx.y) * THD_N;

  float sC[THD_N][THD_N], sA[THD_N], sB[THD_N];

  // set sC to 0
  for (auto i = 0; i < THD_N; i++) {
    for (auto j = 0; j < THD_N; j++) {
      sC[i][j] = 0.0f;
    }
  }

  for (auto k = 0; k < N; k++) {
    // load sA and sB
    for (auto i = 0; i < THD_N; i++) {
      sA[i] = at(A, tx_beg + i, k);
    }
    for (auto j = 0; j < THD_N; j++) {
      sB[j] = at(B, k, ty_beg + j);
    }

    // compute sC
    for (auto i = 0; i < THD_N; i++) {
      for (auto j = 0; j < THD_N; j++) {
        sC[i][j] += sA[i] * sB[j];
      }
    }
  }

  // write back to C
  for (auto i = 0; i < THD_N; i++) {
    for (auto j = 0; j < THD_N; j++) {
      at(C, tx_beg + i, ty_beg + j) = sC[i][j];
    }
  }
}

// host copies of A, B, C
float hA[N * N], hB[N * N], hC[N * N];

int main(void) {
  // init two matrixes
  for (auto i = 0; i < N; i++) {
    for (auto j = 0; j < N; j++) {
      at(hA, i, j) = i == j ? 1.0f : 0.0f;
    }
  }

  for (auto i = 0; i < N; i++) {
    for (auto j = 0; j < N; j++) {
      at(hB, i, j) = i == j ? 1.0f : 0.0f;
    }
  }

  // device copies of A, B, C
  float *A, *B, *C;

  // Allocate Unified Memory – accessible from CPU or GPU
  auto constexpr MAT_SIZE = N * N * sizeof(float);
  hipMallocManaged(&A, MAT_SIZE);
  hipMallocManaged(&B, MAT_SIZE);
  hipMallocManaged(&C, MAT_SIZE);

  hipMemcpy(A, hA, MAT_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(B, hB, MAT_SIZE, hipMemcpyHostToDevice);
  hipMemset(C, 1, MAT_SIZE);

  dim3 constexpr blockDim{BLK_N, BLK_N, 1};
  dim3 constexpr gridDim{N / BLK_N / THD_N, N / BLK_N / THD_N, 1};

  auto const matmul_begin = std::chrono::steady_clock::now();
  matmul<<<gridDim, blockDim>>>(C, A, B);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  auto const matmul_end = std::chrono::steady_clock::now();

  hipMemcpy(hC, C, MAT_SIZE, hipMemcpyDeviceToHost);

  // Check for errors
  for (auto i = 0; i < N; i++) {
    for (auto j = 0; j < N; j++) {
      auto const v = at(C, i, j);
      if ((i == j && !feq(v, 1.0f)) || (i != j && !feq(v, 0.0f))) {
        std::cout << "Error at (" << i << ", " << j << "): " << v << std::endl;
      }
    }
  }

  auto const matmul_time =
      std::chrono::duration_cast<std::chrono::milliseconds>(matmul_end -
                                                            matmul_begin)
          .count();

  // each cell needs N fma, and there are N * N cells
  auto const matmul_tflops = 2.0 * N * N * N / matmul_time / 1e9;
  auto constexpr theory_max_tflops = 14.7456;

  std::cout << std::setprecision(3) << std::fixed;
  std::cout << "matmul time: " << matmul_time << "ms" << std::endl;
  std::cout << "Throughput: " << matmul_tflops << " TFLOPS "
            << "(" << matmul_tflops / theory_max_tflops * 100 << "%)"
            << std::endl;

  // Free memory
  hipFree(A);
  hipFree(B);
  hipFree(C);

  return 0;
}
